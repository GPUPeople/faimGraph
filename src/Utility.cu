//------------------------------------------------------------------------------
// Utility.cu
//
// faimGraph
//
//------------------------------------------------------------------------------
//

#include "Utility.h"

//------------------------------------------------------------------------------
void start_clock(hipEvent_t &start, hipEvent_t &end)
{
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));
    HANDLE_ERROR(hipEventRecord(start,0));
}
//------------------------------------------------------------------------------
float end_clock(hipEvent_t &start, hipEvent_t &end)
{
    float time;
    HANDLE_ERROR(hipEventRecord(end,0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(&time,start,end));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));

    // Returns ms
    return time;
}