#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// PageRank.cu
//
// faimGraph
//
//------------------------------------------------------------------------------
//
#include <thrust/device_vector.h>

#include "MemoryManager.h"
#include "PageRank.h"

//------------------------------------------------------------------------------
// Device funtionality
//------------------------------------------------------------------------------
//

//------------------------------------------------------------------------------
//
template <typename VertexDataType, typename EdgeDataType>
__global__ void d_algPageRankNaive(MemoryManager* memory_manager,
                                   memory_t* memory,
                                   int page_size,
                                   float* page_rank,
                                   float* next_page_rank)
{
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid >= memory_manager->next_free_vertex_index)
    return;

  // PageRank
  VertexDataType* vertices = (VertexDataType*)memory;

  if (vertices[tid].host_identifier == DELETIONMARKER)
    return;

  AdjacencyIterator<EdgeDataType> adjacency_iterator(pageAccess<EdgeDataType>(memory, vertices[tid].mem_index, page_size, memory_manager->start_index));

  float page_factor = page_rank[tid] / vertices[tid].neighbours;

  for (int i = 0; i < vertices[tid].neighbours; ++i)
  {
    atomicAdd(&next_page_rank[adjacency_iterator.getDestination()], page_factor);
    adjacency_iterator.advanceIterator(i, memory_manager->edges_per_page, memory, page_size, memory_manager->start_index);
  }

  return;
}


#define MULTIPLICATOR 4
#define WORKER_SIZE 4
//------------------------------------------------------------------------------
//
template <typename VertexDataType, typename EdgeDataType>
__global__ void d_algPageRankNaiveWarp(MemoryManager* memory_manager,
                                      memory_t* memory,
                                      int page_size,
                                      float* page_rank,
                                      float* next_page_rank)
{
  int warpID = threadIdx.x / WARPSIZE;
  int wid = (blockIdx.x * MULTIPLICATOR) + warpID;  
  int threadID = threadIdx.x - (warpID * WARPSIZE);
  vertex_t edges_per_page = memory_manager->edges_per_page;
  // Outside threads per block (because of indexing structure we use 31 threads)
	if ((threadID >= edges_per_page) || (wid >= memory_manager->next_free_vertex_index))
		return;

	VertexDataType* vertices = (VertexDataType*)memory;
	// PageRank
	__shared__ AdjacencyIterator<EdgeDataType> adjacency_iterator[MULTIPLICATOR];
	__shared__ int neighbours[MULTIPLICATOR], capacity[MULTIPLICATOR];
  

	if (vertices[wid].host_identifier == DELETIONMARKER)
		return;

	if (SINGLE_THREAD_MULTI)
	{
		VertexDataType vertex = vertices[wid];
		adjacency_iterator[warpID].setIterator(pageAccess<EdgeDataType>(memory, vertex.mem_index, page_size, memory_manager->start_index));
		neighbours[warpID] = vertex.neighbours;
    capacity[warpID] = vertex.capacity;
	}
	__syncwarp();

	float page_factor = page_rank[wid] / vertices[wid].neighbours;

	int round = 0;
  while (round < capacity[warpID])
  {
		if(round + threadID < neighbours[warpID])
			atomicAdd(&next_page_rank[adjacency_iterator[warpID].getDestinationAt(threadID)], page_factor);

		round += (edges_per_page);
    // ################ SYNC ################
    __syncwarp();
    // ################ SYNC ################
    if (SINGLE_THREAD_MULTI && round < capacity[warpID])
    {
      // First move adjacency to the last element = index of next block
      adjacency_iterator[warpID].blockTraversalAbsolute(edges_per_page, memory, page_size, memory_manager->start_index);
    }
    // Sync so that everythread has the correct adjacencylist
    // ################ SYNC ################
    __syncwarp();
    // ################ SYNC ################
	}

  return;
}

//------------------------------------------------------------------------------
//
template <typename VertexDataType, typename EdgeDataType>
__global__ void d_algPageRankBalancedWarp(MemoryManager* memory_manager,
																					memory_t* memory,
																					int page_size,
																					float* page_rank,
																					float* next_page_rank,
																					vertex_t* vertex_index,
																					vertex_t* page_per_vertex_index,
																					int page_count)
{
	int warpID = threadIdx.x / WARPSIZE;
  int wid = (blockIdx.x * MULTIPLICATOR) + warpID;  
  int threadID = threadIdx.x - (warpID * WARPSIZE);
  vertex_t edges_per_page = memory_manager->edges_per_page;
  // Outside threads per block (because of indexing structure we use 31 threads)
	if ((threadID >= edges_per_page) || (wid >= page_count))
		return;

	VertexDataType* vertices = (VertexDataType*)memory;
  vertex_t index = vertex_index[wid];
  vertex_t page_index = page_per_vertex_index[wid];
	VertexDataType vertex = vertices[index];

  if (vertices[index].host_identifier == DELETIONMARKER)
		return;

	AdjacencyIterator<EdgeDataType> adjacency_iterator(pageAccess<EdgeDataType>(memory, vertex.mem_index, page_size, memory_manager->start_index));
	float page_factor = page_rank[index] / vertex.neighbours;
  unsigned int iterations{edges_per_page};
  for (int i = page_index; i > 0; --i)
  {
    adjacency_iterator.blockTraversalAbsolute(edges_per_page, memory, page_size, memory_manager->start_index);
  }
  if ((vertex.neighbours) < ((page_index + 1) * edges_per_page))
  {
    iterations = (vertex.neighbours) % edges_per_page;
  }

  page_factor = page_rank[index] / vertex.neighbours;

	if(threadID < iterations)
	{
		atomicAdd(&next_page_rank[adjacency_iterator.getDestinationAt(threadID)], page_factor);
	}

  return;
}

//------------------------------------------------------------------------------
//
template <typename VertexDataType, typename EdgeDataType>
__global__ void d_algPageRankBalanced(MemoryManager* memory_manager,
                                      memory_t* memory,
                                      int page_size,
                                      float* page_rank,
                                      float* next_page_rank,
                                      vertex_t* vertex_index,
                                      vertex_t* page_per_vertex_index,
                                      int page_count)
{
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid >= page_count)
    return;

  // PageRank
  VertexDataType* vertices = (VertexDataType*)memory;
  vertex_t index = vertex_index[tid];
  vertex_t page_index = page_per_vertex_index[tid];
  vertex_t edges_per_page = memory_manager->edges_per_page;
  vertex_t neighbours = vertices[index].neighbours;

  if (vertices[index].host_identifier == DELETIONMARKER)
    return;

  AdjacencyIterator<EdgeDataType> adjacency_iterator(pageAccess<EdgeDataType>(memory, vertices[index].mem_index, page_size, memory_manager->start_index));
  for (int i = page_index; i > 0; --i)
  {
    adjacency_iterator.blockTraversalAbsolute(edges_per_page, memory, page_size, memory_manager->start_index);
  }
  // Now every thread points to its unique page in memory

  int iterations;
  if ((neighbours) < ((page_index + 1) * edges_per_page))
  {
    iterations = (neighbours) % edges_per_page;
  }
  else
  {
    iterations = edges_per_page;
  }

  float page_factor = page_rank[index] / neighbours;

  for (int i = 0; i < iterations; ++i)
  {
    atomicAdd(&next_page_rank[adjacency_iterator.getDestinationAt(i)], page_factor);
  }

  return;
}

//------------------------------------------------------------------------------
//
__global__ void d_applyPageRank(MemoryManager* memory_manager,
                                float* page_rank,
                                float* next_page_rank,
                                float* absolute_difference,
                                float dampening_factor)
{
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid >= memory_manager->next_free_vertex_index)
    return;

  float abs_diff{0.0f};
  if (dampening_factor <= 0)
  {
    // Use standard formula: PR = sum(PR(x)/N(x))
    absolute_difference[tid] = page_rank[tid] - next_page_rank[tid];
    page_rank[tid] = next_page_rank[tid];
		next_page_rank[tid] = 0.0f;
  }
  else
  {
    // Use formula with dampening: PR = (1 - damp)/N +  d*(sum(PR(x)/N(x)))
    abs_diff = page_rank[tid];
    page_rank[tid] = ((1.0f - dampening_factor) / (memory_manager->number_vertices)) + (dampening_factor * next_page_rank[tid]);
    absolute_difference[tid] = abs_diff - page_rank[tid];
		next_page_rank[tid] = 0.0f;
  }
  return;
}


//------------------------------------------------------------------------------
// Host funtionality
//------------------------------------------------------------------------------
//

//------------------------------------------------------------------------------
//
template <typename VertexDataType, typename EdgeDataType>
float PageRank<VertexDataType, EdgeDataType>::algPageRankNaive(const std::unique_ptr<MemoryManager>& memory_manager)
{
  float absDiff = 0.0f;
  int block_size = 256;
  int grid_size = (memory_manager->next_free_vertex_index / block_size) + 1;

	d_algPageRankNaive <VertexDataType, EdgeDataType> << <grid_size, block_size >> > ((MemoryManager*)memory_manager->d_memory,
																																										memory_manager->d_data, 
																																										memory_manager->page_size,
																																										d_page_rank,
																																										d_next_page_rank);


	// block_size = WARPSIZE * MULTIPLICATOR;
	// grid_size = (memory_manager->next_free_vertex_index / MULTIPLICATOR) + 1;
  // d_algPageRankNaiveWarp <VertexDataType, EdgeDataType> << <grid_size, block_size >> > ((MemoryManager*)memory_manager->d_memory,
  //                                                                                    memory_manager->d_data, 
  //                                                                                    memory_manager->page_size,
  //                                                                                    d_page_rank,
  //                                                                                    d_next_page_rank);

	block_size = 256;
	grid_size = (memory_manager->next_free_vertex_index / block_size) + 1;
  // Now we have to set the pagerank
  d_applyPageRank << < grid_size, block_size >> > ((MemoryManager*)memory_manager->d_memory,
                                                    d_page_rank,
                                                    d_next_page_rank,
                                                    d_absolute_difference,
                                                    dampening_factor);


  thrust::device_ptr<float> th_abs_diff(d_absolute_difference);
  thrust::device_ptr<float> th_diff_sum(d_diff_sum);
	thrust::inclusive_scan(th_abs_diff, th_abs_diff + memory_manager->next_free_vertex_index, th_diff_sum);

  // Copy result back to host
  HANDLE_ERROR(hipMemcpy(&absDiff,
                          d_diff_sum + (memory_manager->next_free_vertex_index - 1),
                          sizeof(float),
                          hipMemcpyDeviceToHost));

  return absDiff;
}

template float PageRank<VertexData, EdgeData>::algPageRankNaive (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataWeight, EdgeDataWeight>::algPageRankNaive (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataSemantic, EdgeDataSemantic>::algPageRankNaive (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexData, EdgeDataSOA>::algPageRankNaive (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataWeight, EdgeDataWeightSOA>::algPageRankNaive (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataSemantic, EdgeDataSemanticSOA>::algPageRankNaive (const std::unique_ptr<MemoryManager>& memory_manager);

//------------------------------------------------------------------------------
//
template <typename VertexDataType, typename EdgeDataType>
float PageRank<VertexDataType, EdgeDataType>::algPageRankBalanced(const std::unique_ptr<MemoryManager>& memory_manager)
{
  float absDiff = 0.0f;
  int block_size = 128;
  int grid_size = (memory_manager->next_free_vertex_index / block_size) + 1;

	static int number_pages{0};
  if (d_vertex_index == nullptr)
  {
		number_pages = memory_manager->numberPagesInMemory<VertexDataType>(d_page_count, d_accumulated_page_count);
    TemporaryMemoryAccessHeap temp_memory_dispenser(memory_manager.get(), reinterpret_cast<memory_t*>(d_page_count + memory_manager->next_free_vertex_index + 1));
    d_vertex_index = temp_memory_dispenser.getTemporaryMemory<vertex_t>(number_pages);
    d_page_per_vertex_index = temp_memory_dispenser.getTemporaryMemory<vertex_t>(number_pages);
		memory_manager->workBalanceCalculation(d_accumulated_page_count, number_pages, d_vertex_index, d_page_per_vertex_index);
  }

  grid_size = (number_pages / block_size) + 1;
  d_algPageRankBalanced <VertexDataType, EdgeDataType> << <grid_size, block_size >> > ((MemoryManager*)memory_manager->d_memory,
                                                                                        memory_manager->d_data,
                                                                                        memory_manager->page_size,
                                                                                        d_page_rank,
                                                                                        d_next_page_rank,
                                                                                        d_vertex_index,
                                                                                        d_page_per_vertex_index,
																																												number_pages);
																																												
	// block_size = WARPSIZE * MULTIPLICATOR;
	// grid_size = (number_pages / MULTIPLICATOR) + 1;
	// d_algPageRankBalancedWarp <VertexDataType, EdgeDataType> << <grid_size, block_size >> > ((MemoryManager*)memory_manager->d_memory,
  //                                                                                       memory_manager->d_data,
  //                                                                                       memory_manager->page_size,
  //                                                                                       d_page_rank,
  //                                                                                       d_next_page_rank,
  //                                                                                       d_vertex_index,
  //                                                                                       d_page_per_vertex_index,
	// 																																											number_pages);

  grid_size = (memory_manager->next_free_vertex_index / block_size) + 1;

  // Now we have to set the pagerank
  d_applyPageRank << < grid_size, block_size >> > ((MemoryManager*)memory_manager->d_memory,
                                                    d_page_rank,
                                                    d_next_page_rank,
                                                    d_absolute_difference,
                                                    dampening_factor);


  thrust::device_ptr<float> th_abs_diff(d_absolute_difference);
  thrust::device_ptr<float> th_diff_sum(d_diff_sum);
  thrust::inclusive_scan(th_abs_diff, th_abs_diff + memory_manager->next_free_vertex_index, th_diff_sum);

  // Copy result back to host
  HANDLE_ERROR(hipMemcpy(&absDiff,
                          d_diff_sum + (memory_manager->next_free_vertex_index - 1),
                          sizeof(float),
                          hipMemcpyDeviceToHost));

  return absDiff;
}

template float PageRank<VertexData, EdgeData>::algPageRankBalanced (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataWeight, EdgeDataWeight>::algPageRankBalanced (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataSemantic, EdgeDataSemantic>::algPageRankBalanced (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexData, EdgeDataSOA>::algPageRankBalanced (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataWeight, EdgeDataWeightSOA>::algPageRankBalanced (const std::unique_ptr<MemoryManager>& memory_manager);
template float PageRank<VertexDataSemantic, EdgeDataSemanticSOA>::algPageRankBalanced (const std::unique_ptr<MemoryManager>& memory_manager);